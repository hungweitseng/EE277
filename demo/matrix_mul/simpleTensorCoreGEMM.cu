/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <sys/time.h>
#include <pthread.h>
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
/*#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384
#define MATRIX_M 10240
#define MATRIX_N 10240
#define MATRIX_K 10240
*/
/*#define MATRIX_M 2048
#define MATRIX_N 2048
#define MATRIX_K 2048*/



// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
   // Leading dimensions. Packed with no transpositions.
   int lda = M;
   int ldb = K;
   int ldc = M;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < K; i += WMMA_K) {
      int aRow = warpM * WMMA_M;
      int aCol = i;

      int bRow = i;
      int bCol = warpN * WMMA_N;

      // Bounds checking
      if (aRow < M && aCol < K && bRow < K && bCol < N) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < M && cCol < N) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);


      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n, float scale) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      if(scale > 32768.0)
          out[idx] = (half)(in[idx]/scale);
      else
      out[idx] = in[idx];
   }
}

__global__ void scaleRandom (float *out, float *in, int n, float scale) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
          out[idx] = (int)(in[idx]*scale);
   }
}

void* cublasCreateThread(void *x)
{
//   struct timeval time_start, time_end, total_start, total_end;
//   int init_time;
   hipblasHandle_t* cublasHandle = (hipblasHandle_t *)x;
//    gettimeofday(&time_start, NULL);
   cublasErrCheck(hipblasCreate(cublasHandle));
//   cublasErrCheck(cublasSetMathMode(*cublasHandle, CUBLAS_TENSOR_OP_MATH));
//	fprintf(stderr,"cublasCreate\n");
//  	gettimeofday(&time_end, NULL);
//        init_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
//	fprintf(stderr,"cublasCreate  %d\n",init_time);
  return NULL;
}
void* cublasCreateDefaultThread(void *x)
{
//   struct timeval time_start, time_end, total_start, total_end;
//   int init_time;
   hipblasHandle_t* cublasHandle = (hipblasHandle_t *)x;
//    gettimeofday(&time_start, NULL);
   cublasErrCheck(hipblasCreate(cublasHandle));
//   cublasErrCheck(cublasSetMathMode(*cublasHandle, CUBLAS_DEFAULT_MATH));
//  	gettimeofday(&time_end, NULL);
//        init_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
//	fprintf(stderr,"cublasCreate (2) %d\n",init_time);
  return NULL;
}

int main(int argc, char* argv[]) {
   float *a_fp32;
   float *b_fp32;
   float scale;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas;
   float *c_wmma;
   float *c_sgemm;
   float *c_cublas_gemmEx;

   float *c_host_cublas;
   float *c_host_cublasCublasGemmEx;
   float *c_host_wmma;
   float *c_host_sgemm;

   float alpha = 1.0f;
   float beta = 0.0f;

int MATRIX_M=10240;
int MATRIX_N=10240;
int MATRIX_K=10240;
   
   hiprandGenerator_t gen;
   hipblasHandle_t cublasHandle;
   hipblasHandle_t cublasHandle_default;
   
   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   
   hipEvent_t startcublasEX;
   hipEvent_t stopcublasEX;

   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   hipEvent_t startcublasCublasGemmEx;
   hipEvent_t stopcublasCublasGemmEx;
    struct timeval time_start, time_end, total_start, total_end;
   int init_time;
    gettimeofday(&time_start, NULL);
    hipFree(0);
   #ifdef THREAD
  pthread_t thread;
  pthread_t thread1;
  pthread_create(&thread, NULL, cublasCreateThread, &cublasHandle);
  pthread_create(&thread1, NULL, cublasCreateDefaultThread, &cublasHandle_default);
   #endif  
   cudaErrCheck(hipEventCreate(&startWMMA));
   cudaErrCheck(hipEventCreate(&stopWMMA));
   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));

   cudaErrCheck(hipEventCreate(&startcublasEX));
   cudaErrCheck(hipEventCreate(&stopcublasEX));

   cudaErrCheck(hipEventCreate(&startcublasCublasGemmEx));
   cudaErrCheck(hipEventCreate(&stopcublasCublasGemmEx));
   
   #ifndef THREAD
    gettimeofday(&total_start, NULL);
   cublasErrCheck(hipblasCreate(&cublasHandle));
   cublasErrCheck(hipblasCreate(&cublasHandle_default));
   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
   cublasErrCheck(hipblasSetMathMode(cublasHandle_default, HIPBLAS_DEFAULT_MATH));
  	gettimeofday(&total_end, NULL);
        init_time = ((total_end.tv_sec * 1000000 + total_end.tv_usec) - (total_start.tv_sec * 1000000 + total_start.tv_usec));
	fprintf(stderr,"cublasCreate(2)  %d\n",init_time);
   #endif
   MATRIX_M = atoi(argv[1]);
   MATRIX_K = MATRIX_M;
   MATRIX_N = MATRIX_M;
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas_gemmEx, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_sgemm, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_cublasCublasGemmEx = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_sgemm = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

   curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));
   scale = atof(argv[2]);
   if(scale == 0.0)
       scale=3.4028234664e+38;
   printf("Scale: %f\n",scale);
   scaleRandom <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp32, a_fp32, MATRIX_M * MATRIX_K, scale);
   scaleRandom <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (b_fp32, b_fp32, MATRIX_N * MATRIX_K, scale);
   
   // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K, scale);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N, scale);

   curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));
   
   curandErrCheck(hiprandDestroyGenerator(gen));
   
   cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   cudaErrCheck(hipMemcpy(c_cublas_gemmEx, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
#ifdef THREAD
  pthread_join(thread, NULL); 
  pthread_join(thread1, NULL); 
//   cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));
   cublasErrCheck(cublasSetMathMode(cublasHandle_default, CUBLAS_DEFAULT_MATH));
#endif
  	gettimeofday(&time_end, NULL);
        init_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec));
	fprintf(stderr,"Before GEMM %d\n",init_time);


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

   printf("Running with sgemm...\n");
   cudaErrCheck(hipEventRecord(startcublas));
   hipblasSgemm(cublasHandle_default, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_M, MATRIX_N, MATRIX_K, &alpha, a_fp32, MATRIX_M, b_fp32, MATRIX_N, &beta, c_sgemm, MATRIX_K);
   cudaErrCheck(hipEventRecord(stopcublas));
   
   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;
 
   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);


   
   printf("Running with wmma...\n");
   cudaErrCheck(hipEventRecord(startWMMA));
   wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   cudaErrCheck(hipEventRecord(stopWMMA));


   // Now using cuBLAS but not tensor
   printf("Running with cuBLAS (GemmEX)...\n");
   cudaErrCheck(hipEventRecord(startcublasCublasGemmEx));
   cublasErrCheck(hipblasGemmEx(cublasHandle_default, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas_gemmEx, HIP_R_32F, MATRIX_M,
                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
   cudaErrCheck(hipEventRecord(stopcublasCublasGemmEx));

//   cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));
   
   // Now using cuBLAS
   printf("Running with cuBLAS...\n");
   cudaErrCheck(hipEventRecord(startcublasEX));
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   cudaErrCheck(hipEventRecord(stopcublasEX));

   // Error checking
   cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_cublasCublasGemmEx, c_cublas_gemmEx, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_sgemm, c_sgemm, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));

   printf("\nChecking results with cublas (cublasGemmEx)...\n");
   int errors_default = 0;
   double error_rate =0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
      float v1 = c_host_cublasCublasGemmEx[i];
      float v2 = c_host_sgemm[i];
      if(scale > 32768)
          v1 = v1*scale*scale;
      //    MATRIX_M * MATRIX_N;
      error_rate += (abs(v1 - v2)/v1);
      if (abs(v1 - v2) > 1e-5) {
//      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-3) {
         errors_default++;
         if (errors_default < 3) printf("%f %f\n", v1, v2);
      }
   }
   if (errors_default > 0) {
      printf("GemmEX does not agree with cuBLAS default! %d errors -- error rate %lf!\n", errors_default/MATRIX_M * MATRIX_N,error_rate/errors_default);
   }
   
   printf("\nChecking results with tensor cores...\n");
   // 0.01% relative tolerance. 1e-5 absolute tolerance.
   int errors = 0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
      float v1 = c_host_wmma[i];
      float v2 = c_host_sgemm[i];
//      float v2 = c_host_cublasCublasGemmEx[i];
      error_rate += (abs(v1 - v2)/v1);
//      float v2 = c_host_cublasCublasGemmEx[i];
      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-3) {
         errors++;
         if (errors < 3) printf("%f %f\n", v1, v2);
      }
   }

   if (errors > 0) {
      printf("WMMA does not agree with cuBLAS! %d errors!-- error rate %lf!\n", errors_default,error_rate/errors_default);
   }
   errors = 0;
   error_rate =0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
//      float v1 = c_host_sgemm[i];
      float v1 = c_host_cublas[i];
      float v2 = c_host_sgemm[i];
     if(scale > 32768)
          v1 = v1*scale*scale;
 //      float v2 = c_host_cublasCublasGemmEx[i];
      error_rate += (abs(v1 - v2)/v1);
//      float v2 = c_host_cublasCublasGemmEx[i];
      if (abs(v1 - v2) > 1e-5) {
//      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-3) {
         errors++;
         if (errors < 3) printf("%f %f\n", v1, v2);
      }
   }

   if (errors > 0) {
      printf("cuBLAS TCU not agree with cuBLAS! %.10lf errors!-- error rate %.10lf!\n", (float)errors/(MATRIX_M * MATRIX_N),error_rate/errors);
   }

//   else {
//   {
//      printf("Results verified: cublas and WMMA agree.\n\n");
      float wmmaTime;
      float cublasTime;
      cudaErrCheck(hipEventSynchronize(stopWMMA));
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("wmma took %fms\n", wmmaTime);
      printf("cublas took %fms\n", cublasTime);
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublasCublasGemmEx, stopcublasCublasGemmEx));
      printf("cublas hipblasGemmEx took %fms\n", cublasTime);
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublasEX, stopcublasEX));
      printf("cublas tensor cores took %fms\n", cublasTime);

      printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
//   }
   printf("Running with cuBLASCdot...\n");
   hipEvent_t startDot;
   hipEvent_t stopDot;
   cudaErrCheck(hipEventCreate(&startDot));
   cudaErrCheck(hipEventCreate(&stopDot));

   hipblasSetPointerMode(cublasHandle_default,HIPBLAS_POINTER_MODE_DEVICE); // set here!!!
   cudaErrCheck(hipEventRecord(startDot));
   for(int i = 0; i< MATRIX_M; i++)
   {
       cublasErrCheck(hipblasSdot(cublasHandle_default,MATRIX_N,&a_fp32[i*MATRIX_N],1,&b_fp32[i*MATRIX_N],1,&c_cublas_gemmEx[i*MATRIX_N]));
   }
   cudaErrCheck(hipEventRecord(stopDot));
      cudaErrCheck(hipEventSynchronize(stopDot));

   cudaErrCheck(hipEventElapsedTime(&cublasTime, startDot, stopDot));
   printf("cublas dot product took %fms\n", cublasTime);
   
   
   cudaErrCheck(hipEventDestroy(startWMMA));
   cudaErrCheck(hipEventDestroy(stopWMMA));

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas));
   cudaErrCheck(hipFree(c_wmma));
   
   free(c_host_cublas);
   free(c_host_wmma);

   cudaErrCheck(hipDeviceReset());
   return 0;
}


